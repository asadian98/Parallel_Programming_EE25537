#include "hip/hip_runtime.h"
//ONLY MODIFY THIS FILE!
//YOU CAN MODIFY EVERYTHING IN THIS FILE!

#include "bmm.h"

#define tx threadIdx.x
#define ty threadIdx.y
#define tz threadIdx.z

#define bx blockIdx.x
#define by blockIdx.y
#define bz blockIdx.z

// TILEX and TILEY are used to set the number of threads in a CUDA block 
#define TILEX 32
#define TILEY 16

// you may define other parameters here!
#define TM (TILEY < TILEX ? TILEY : TILEX)
// you may define other macros here!
// you may define other functions here!

dim3 getDimGrid(const int m, const int n) {
	dim3 dimGrid(n/TILEX,n/TILEY);
	return dimGrid;
}
dim3 getDimBlock(const int m, const int n) {
	dim3 dimBlock(TILEX,TILEY);
	return dimBlock;
}

__global__ void kernelFunc(float* ad, float* bd, float* cd, const int m, const int n) {
	
	// write your GPU kernel function here
	
	__shared__ float as[TILEY][2*TM];
	__shared__ float bs[2*TM][TILEX];

	int i = TILEY*by + ty;
	int j = TILEX*bx + tx;
	//int tmp = 0;
	float s = 0;
	int k;
	int r = 0;
	int offset;
	int offsetx;
	int offsety;
	int im = (i << m);
	for (; r < n/(2*TM); r++){
		//init as,bs:
		offset = TM * 2 * r;
		offsetx = im + offset + tx;
		offsety = offset + ty;
		if(tx < TM){
			as[ty][tx] = ad[offsetx];
			as[ty][tx + TM] = ad[offsetx + TM];
		}
		if(ty < TM){
			bs[ty][tx] = bd[((offsety) << m) + j];
			bs[ty + TM][tx] = bd[((offsety + TM) << m) + j];
		}
		//tmp = tmp + 4*TM;
		__syncthreads();

		for (k = 0; k < 2*TM; k++)
			//use as,bs:
			s += as[ty][k] * bs[k][tx];
		
		__syncthreads();
	}
	cd[(i << m)+ j] = s;
}