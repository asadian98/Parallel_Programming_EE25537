#include "hip/hip_runtime.h"
// ONLY MODIFY THIS FILE

#include "scan2.h"
#include "gpuerrors.h"

#define tx threadIdx.x
#define ty threadIdx.y
#define tz threadIdx.z

#define bx blockIdx.x
#define by blockIdx.y
#define bz blockIdx.z

// you may define other parameters here!
#define TILEX 1024

// you may define other macros here!
// you may define other functions here!

__global__ void kernelFunc1(float* a, float* w)
{
	__shared__ float as[TILEX];

    int i = 1024 * bx + tx;

    as[tx] = a[i];

    __syncthreads();
	
	int j;
	int j2;
	if(tx == 1024 - 1) as[tx] = 0;
	
	for(j = 1; j < 10; j += 1){
		j2 = 1 << j;
        if((tx % j2 == j2 - 1) && tx != 1024 - 1){
            as[tx] = as[tx] + as[tx - (j2>>1)];
        }
        __syncthreads();
    }
	
	float as_tmp;
	int offset;
	
    for(; j >= 1; j -= 1){
		j2 = 1 << j;
        offset = tx - (j2>>1);
		if(tx % j2 == j2 - 1){
            as_tmp = as[offset];
            as[offset] = as[tx];
            as[tx] = as[tx] + as_tmp;
        }
        __syncthreads();
    }	
	
	if(tx == (1024 - 1)){
        w[bx] = a[i] + as[tx];
    }

	a[i] = as[tx];
}

__global__ void kernelFunc3(float* a, int blockSize)
{
	__shared__ float as[TILEX];
	__shared__ float bs[TILEX];

    int i = blockSize * bx + tx;

	as[tx] = (tx > 0) ? a[i-1] : 0;

    __syncthreads();
	
	int j = 2;
	int k = 0;
	int offset;
	
	for(; j <= blockSize; j <<= 1){
		k = 1 - k;
		offset = tx - (j>>1);
		if(tx >= (j>>1))
			if(k) bs[tx] = as[tx] + as[offset];
			else as[tx] = bs[tx] + bs[offset];
		else
			if(k) bs[tx] = as[tx];
			else as[tx] = bs[tx];
		__syncthreads();
    }
	if(k) a[i] = bs[tx];
	else a[i] = as[tx];
}

__global__ void kernelFunc2(float* a, float* b)
{
	int i = 1024 * bx + tx;
    a[i] = b[bx] + a[i];
}

void gpuKernel(float* a, float* c,int n) {

	if(n == (1 << 20)){
		
		float* ad;
		float* w1;
	
		hipMalloc((void**)&ad, n*sizeof(float));
		hipMalloc((void**)&w1, (n>>10)*sizeof(float));
		
		hipMemcpy(ad, a, n*sizeof(float), hipMemcpyHostToDevice);
		
		kernelFunc1<<< n>>10 , 1024 >>>(ad,w1);
		kernelFunc3<<< 1 , n>>10 >>> (w1, 1024);
		
		kernelFunc2<<< n>>10 , 1024 >>> (ad,w1);
		
		hipMemcpy(c, ad+1, (n-1)*sizeof(float), hipMemcpyDeviceToHost);
		c[n-1] = c[n-2] + a[n-1];
		
		hipFree(ad);
		hipFree(w1);
	}
	else if(n <= (1 << 25)){
		float* ad;
		float* w1;
		float* w2;
	
		hipMalloc((void**)&ad, n*sizeof(float));
		hipMalloc((void**)&w1, (n>>10)*sizeof(float));
		hipMalloc((void**)&w2, (n>>20)*sizeof(float));
		
		hipMemcpy(ad, a, n*sizeof(float), hipMemcpyHostToDevice);
		
		kernelFunc1<<< n>>10 , 1024 >>>(ad,w1);
		kernelFunc1<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc3<<< 1 , n>>20 >>> (w2, n>>20);
		
		kernelFunc2<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc2<<< n>>10 , 1024 >>> (ad,w1);
		
		hipMemcpy(c, ad+1, (n-1)*sizeof(float), hipMemcpyDeviceToHost);
		c[n-1] = c[n-2] + a[n-1];
		
		hipFree(ad);
		hipFree(w1);
		hipFree(w2);
	}
	else if(n == (1 << 26)){
		
		n = n / 2;
		
		float* ad;
		float* w1;
		float* w2;
	
		hipMalloc((void**)&ad, n*sizeof(float));
		hipMalloc((void**)&w1, (n>>10)*sizeof(float));
		hipMalloc((void**)&w2, (n>>20)*sizeof(float));
		
		hipMemcpy(ad, a, n*sizeof(float), hipMemcpyHostToDevice);
		
		kernelFunc1<<< n>>10 , 1024 >>>(ad,w1);
		kernelFunc1<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc3<<< 1 , n>>20 >>> (w2, n>>20);
		
		kernelFunc2<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc2<<< n>>10 , 1024 >>> (ad,w1);
		
		hipMemcpy(c, ad+1, (n-1)*sizeof(float), hipMemcpyDeviceToHost);
		c[n-1] = c[n-2] + a[n-1];
		a[n-1] = c[n-1];
		
		hipMemcpy(ad, a+n-1, n*sizeof(float), hipMemcpyHostToDevice);
		
		kernelFunc1<<< n>>10 , 1024 >>>(ad,w1);
		kernelFunc1<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc3<<< 1 ,n>>20 >>> (w2, n>>20);
		
		kernelFunc2<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc2<<< n>>10 , 1024 >>> (ad,w1);
		
		hipMemcpy(c+n, ad+2, (n-2)*sizeof(float), hipMemcpyDeviceToHost);
		c[2*n-2] = c[2*n-3] + a[2*n-2];
		c[2*n-1] = c[2*n-2] + a[2*n-1];
		
		hipFree(ad);
		hipFree(w1);
		hipFree(w2);
	}
	else if(n == (1 << 27)){
		
		n = n / 4;
		
		float* ad;
		float* w1;
		float* w2;
	
		hipMalloc((void**)&ad, n*sizeof(float));
		hipMalloc((void**)&w1, (n>>10)*sizeof(float));
		hipMalloc((void**)&w2, (n>>20)*sizeof(float));
		
		hipMemcpy(ad, a, n*sizeof(float), hipMemcpyHostToDevice);
		
		kernelFunc1<<< n>>10 , 1024 >>>(ad,w1);
		kernelFunc1<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc3<<< 1 , n>>20 >>> (w2, n>>20);
		
		kernelFunc2<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc2<<< n>>10 , 1024 >>> (ad,w1);
		
		hipMemcpy(c, ad+1, (n-1)*sizeof(float), hipMemcpyDeviceToHost);
		c[n-1] = c[n-2] + a[n-1];
		a[n-1] = c[n-1];
		
		hipMemcpy(ad, a+n-1, n*sizeof(float), hipMemcpyHostToDevice);
		
		kernelFunc1<<< n>>10 , 1024 >>>(ad,w1);
		kernelFunc1<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc3<<< 1 , n>>20 >>> (w2, n>>20);
		
		kernelFunc2<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc2<<< n>>10 , 1024 >>> (ad,w1);
		
		hipMemcpy(c+n, ad+2, (n-2)*sizeof(float), hipMemcpyDeviceToHost);
		c[2*n-2] = c[2*n-3] + a[2*n-2];
		c[2*n-1] = c[2*n-2] + a[2*n-1];
		a[2*n-1] = c[2*n-1];
		
		hipMemcpy(ad, a+2*n-1, n*sizeof(float), hipMemcpyHostToDevice);
		
		kernelFunc1<<< n>>10 , 1024 >>>(ad,w1);
		kernelFunc1<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc3<<< 1 , n>>20 >>> (w2, n>>20);
		
		kernelFunc2<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc2<<< n>>10 , 1024 >>> (ad,w1);
		
		hipMemcpy(c+2*n, ad+2, (n-2)*sizeof(float), hipMemcpyDeviceToHost);
		c[3*n-2] = c[3*n-3] + a[3*n-2];
		c[3*n-1] = c[3*n-2] + a[3*n-1];
		a[3*n-1] = c[3*n-1];
				
		hipMemcpy(ad, a+3*n-1, n*sizeof(float), hipMemcpyHostToDevice);
		
		kernelFunc1<<< n>>10 , 1024 >>>(ad,w1);
		kernelFunc1<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc3<<< 1 , n>>20 >>> (w2, n>>20);
		
		kernelFunc2<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc2<<< n>>10 , 1024 >>> (ad,w1);
		
		hipMemcpy(c+3*n, ad+2, (n-2)*sizeof(float), hipMemcpyDeviceToHost);
		c[4*n-2] = c[4*n-3] + a[4*n-2];
		c[4*n-1] = c[4*n-2] + a[4*n-1];
		
		hipFree(ad);
		hipFree(w1);
		hipFree(w2);
	}
	else if(n == (1 << 28)){
		
		n = n / 8;
		
		float* ad;
		float* w1;
		float* w2;
	
		hipMalloc((void**)&ad, n*sizeof(float));
		hipMalloc((void**)&w1, (n>>10)*sizeof(float));
		hipMalloc((void**)&w2, (n>>20)*sizeof(float));

		hipMemcpy(ad, a, n*sizeof(float), hipMemcpyHostToDevice);
		
		kernelFunc1<<< n>>10 , 1024 >>>(ad,w1);
		kernelFunc1<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc3<<< 1 , n>>20 >>> (w2, n>>20);
		
		kernelFunc2<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc2<<< n>>10 , 1024 >>> (ad,w1);
		
		hipMemcpy(c, ad+1, (n-1)*sizeof(float), hipMemcpyDeviceToHost);
		c[n-1] = c[n-2] + a[n-1];
		a[n-1] = c[n-1];
		
		hipMemcpy(ad, a+n-1, n*sizeof(float), hipMemcpyHostToDevice);
		
		kernelFunc1<<< n>>10 , 1024 >>>(ad,w1);
		kernelFunc1<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc3<<< 1 , n>>20 >>> (w2, n>>20);
		
		kernelFunc2<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc2<<< n>>10 , 1024 >>> (ad,w1);
		
		hipMemcpy(c+n, ad+2, (n-2)*sizeof(float), hipMemcpyDeviceToHost);
		c[2*n-2] = c[2*n-3] + a[2*n-2];
		c[2*n-1] = c[2*n-2] + a[2*n-1];
		a[2*n-1] = c[2*n-1];
		
		hipMemcpy(ad, a+2*n-1, n*sizeof(float), hipMemcpyHostToDevice);
		
		kernelFunc1<<< n>>10 , 1024 >>>(ad,w1);
		kernelFunc1<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc3<<< 1 , n>>20 >>> (w2, n>>20);
		
		kernelFunc2<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc2<<< n>>10 , 1024 >>> (ad,w1);
		
		hipMemcpy(c+2*n, ad+2, (n-2)*sizeof(float), hipMemcpyDeviceToHost);
		c[3*n-2] = c[3*n-3] + a[3*n-2];
		c[3*n-1] = c[3*n-2] + a[3*n-1];
		a[3*n-1] = c[3*n-1];
				
		hipMemcpy(ad, a+3*n-1, n*sizeof(float), hipMemcpyHostToDevice);
		
		kernelFunc1<<< n>>10 , 1024 >>>(ad,w1);
		kernelFunc1<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc3<<< 1 , n>>20 >>> (w2, n>>20);
		
		kernelFunc2<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc2<<< n>>10 , 1024 >>> (ad,w1);
		
		hipMemcpy(c+3*n, ad+2, (n-2)*sizeof(float), hipMemcpyDeviceToHost);
		c[4*n-2] = c[4*n-3] + a[4*n-2];
		c[4*n-1] = c[4*n-2] + a[4*n-1];
		a[4*n-1] = c[4*n-1];
		
		hipMemcpy(ad, a+4*n-1, n*sizeof(float), hipMemcpyHostToDevice);
		
		kernelFunc1<<< n>>10 , 1024 >>>(ad,w1);
		kernelFunc1<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc3<<< 1 , n>>20 >>> (w2, n>>20);
		
		kernelFunc2<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc2<<< n>>10 , 1024 >>> (ad,w1);
		
		hipMemcpy(c+4*n, ad+2, (n-2)*sizeof(float), hipMemcpyDeviceToHost);
		c[5*n-2] = c[5*n-3] + a[5*n-2];
		c[5*n-1] = c[5*n-2] + a[5*n-1];
		a[5*n-1] = c[5*n-1];
		
		hipMemcpy(ad, a+5*n-1, n*sizeof(float), hipMemcpyHostToDevice);
		
		kernelFunc1<<< n>>10 , 1024 >>>(ad,w1);
		kernelFunc1<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc3<<< 1 , n>>20 >>> (w2, n>>20);
		
		kernelFunc2<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc2<<< n>>10 , 1024 >>> (ad,w1);
		
		hipMemcpy(c+5*n, ad+2, (n-2)*sizeof(float), hipMemcpyDeviceToHost);
		c[6*n-2] = c[6*n-3] + a[6*n-2];
		c[6*n-1] = c[6*n-2] + a[6*n-1];
		a[6*n-1] = c[6*n-1];
		
		hipMemcpy(ad, a+6*n-1, n*sizeof(float), hipMemcpyHostToDevice);
		
		kernelFunc1<<< n>>10 , 1024 >>>(ad,w1);
		kernelFunc1<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc3<<< 1 , n>>20 >>> (w2, n>>20);
		
		kernelFunc2<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc2<<< n>>10 , 1024 >>> (ad,w1);
		
		hipMemcpy(c+6*n, ad+2, (n-2)*sizeof(float), hipMemcpyDeviceToHost);
		c[7*n-2] = c[7*n-3] + a[7*n-2];
		c[7*n-1] = c[7*n-2] + a[7*n-1];
		a[7*n-1] = c[7*n-1];
		
		hipMemcpy(ad, a+7*n-1, n*sizeof(float), hipMemcpyHostToDevice);
		
		kernelFunc1<<< n>>10 , 1024 >>>(ad,w1);
		kernelFunc1<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc3<<< 1 , n>>20 >>> (w2, n>>20);
		
		kernelFunc2<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc2<<< n>>10 , 1024 >>> (ad,w1);
		
		hipMemcpy(c+7*n, ad+2, (n-2)*sizeof(float), hipMemcpyDeviceToHost);
		c[8*n-2] = c[8*n-3] + a[8*n-2];
		c[8*n-1] = c[8*n-2] + a[8*n-1];
		
		
		hipFree(ad);
		hipFree(w1);
		hipFree(w2);
	}
	else if(n == (1 << 29)){
		
		n = n / 16;
		
		float* ad;
		float* w1;
		float* w2;
	
		hipMalloc((void**)&ad, n*sizeof(float));
		hipMalloc((void**)&w1, (n>>10)*sizeof(float));
		hipMalloc((void**)&w2, (n>>20)*sizeof(float));
		
		hipMemcpy(ad, a, n*sizeof(float), hipMemcpyHostToDevice);
		
		kernelFunc1<<< n>>10 , 1024 >>>(ad,w1);
		kernelFunc1<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc3<<< 1 , n>>20 >>> (w2, n>>20);
		
		kernelFunc2<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc2<<< n>>10 , 1024 >>> (ad,w1);
		
		hipMemcpy(c, ad+1, (n-1)*sizeof(float), hipMemcpyDeviceToHost);
		c[n-1] = c[n-2] + a[n-1];
		a[n-1] = c[n-1];
		
		hipMemcpy(ad, a+n-1, n*sizeof(float), hipMemcpyHostToDevice);
		
		kernelFunc1<<< n>>10 , 1024 >>>(ad,w1);
		kernelFunc1<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc3<<< 1 , n>>20 >>> (w2, n>>20);
		
		kernelFunc2<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc2<<< n>>10 , 1024 >>> (ad,w1);
		
		hipMemcpy(c+n, ad+2, (n-2)*sizeof(float), hipMemcpyDeviceToHost);
		c[2*n-2] = c[2*n-3] + a[2*n-2];
		c[2*n-1] = c[2*n-2] + a[2*n-1];
		a[2*n-1] = c[2*n-1];
		
		hipMemcpy(ad, a+2*n-1, n*sizeof(float), hipMemcpyHostToDevice);
		
		kernelFunc1<<< n>>10 , 1024 >>>(ad,w1);
		kernelFunc1<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc3<<< 1 , n>>20>>> (w2, n>>20);
		
		kernelFunc2<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc2<<< n>>10 , 1024 >>> (ad,w1);
		
		hipMemcpy(c+2*n, ad+2, (n-2)*sizeof(float), hipMemcpyDeviceToHost);
		c[3*n-2] = c[3*n-3] + a[3*n-2];
		c[3*n-1] = c[3*n-2] + a[3*n-1];
		a[3*n-1] = c[3*n-1];
				
		hipMemcpy(ad, a+3*n-1, n*sizeof(float), hipMemcpyHostToDevice);
		
		kernelFunc1<<< n>>10 , 1024 >>>(ad,w1);
		kernelFunc1<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc3<<< 1 , n>>20 >>> (w2, n>>20);
		
		kernelFunc2<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc2<<< n>>10 , 1024 >>> (ad,w1);
		
		hipMemcpy(c+3*n, ad+2, (n-2)*sizeof(float), hipMemcpyDeviceToHost);
		c[4*n-2] = c[4*n-3] + a[4*n-2];
		c[4*n-1] = c[4*n-2] + a[4*n-1];
		a[4*n-1] = c[4*n-1];
		
		hipMemcpy(ad, a+4*n-1, n*sizeof(float), hipMemcpyHostToDevice);
		
		kernelFunc1<<< n>>10 , 1024 >>>(ad,w1);
		kernelFunc1<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc3<<< 1 , n>>20 >>> (w2, n>>20);
		
		kernelFunc2<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc2<<< n>>10 , 1024 >>> (ad,w1);
		
		hipMemcpy(c+4*n, ad+2, (n-2)*sizeof(float), hipMemcpyDeviceToHost);
		c[5*n-2] = c[5*n-3] + a[5*n-2];
		c[5*n-1] = c[5*n-2] + a[5*n-1];
		a[5*n-1] = c[5*n-1];
		
		hipMemcpy(ad, a+5*n-1, n*sizeof(float), hipMemcpyHostToDevice);
		
		kernelFunc1<<< n>>10 , 1024 >>>(ad,w1);
		kernelFunc1<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc3<<< 1 , n>>20 >>> (w2, n>>20);
		
		kernelFunc2<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc2<<< n>>10 , 1024 >>> (ad,w1);
		
		hipMemcpy(c+5*n, ad+2, (n-2)*sizeof(float), hipMemcpyDeviceToHost);
		c[6*n-2] = c[6*n-3] + a[6*n-2];
		c[6*n-1] = c[6*n-2] + a[6*n-1];
		a[6*n-1] = c[6*n-1];
		
		hipMemcpy(ad, a+6*n-1, n*sizeof(float), hipMemcpyHostToDevice);
		
		kernelFunc1<<< n>>10 , 1024 >>>(ad,w1);
		kernelFunc1<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc3<<< 1 , n>>20 >>> (w2, n>>20);
		
		kernelFunc2<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc2<<< n>>10 , 1024 >>> (ad,w1);
		
		hipMemcpy(c+6*n, ad+2, (n-2)*sizeof(float), hipMemcpyDeviceToHost);
		c[7*n-2] = c[7*n-3] + a[7*n-2];
		c[7*n-1] = c[7*n-2] + a[7*n-1];
		a[7*n-1] = c[7*n-1];
		
		hipMemcpy(ad, a+7*n-1, n*sizeof(float), hipMemcpyHostToDevice);
		
		kernelFunc1<<< n>>10 , 1024 >>>(ad,w1);
		kernelFunc1<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc3<<< 1 , n>>20 >>> (w2, n>>20);
		
		kernelFunc2<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc2<<< n>>10 , 1024 >>> (ad,w1);
		
		hipMemcpy(c+7*n, ad+2, (n-2)*sizeof(float), hipMemcpyDeviceToHost);
		c[8*n-2] = c[8*n-3] + a[8*n-2];
		c[8*n-1] = c[8*n-2] + a[8*n-1];
		a[8*n-1] = c[8*n-1];
				
		hipMemcpy(ad, a+8*n-1, n*sizeof(float), hipMemcpyHostToDevice);
		
		kernelFunc1<<< n>>10 , 1024 >>>(ad,w1);
		kernelFunc1<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc3<<< 1 , n>>20 >>> (w2, n>>20);
		
		kernelFunc2<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc2<<< n>>10 , 1024 >>> (ad,w1);
		
		hipMemcpy(c+8*n, ad+2, (n-2)*sizeof(float), hipMemcpyDeviceToHost);
		c[9*n-2] = c[9*n-3] + a[9*n-2];
		c[9*n-1] = c[9*n-2] + a[9*n-1];
		a[9*n-1] = c[9*n-1];
				
		hipMemcpy(ad, a+9*n-1, n*sizeof(float), hipMemcpyHostToDevice);
		
		kernelFunc1<<< n>>10 , 1024 >>>(ad,w1);
		kernelFunc1<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc3<<< 1 , n>>20 >>> (w2, n>>20);
		
		kernelFunc2<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc2<<< n>>10 , 1024 >>> (ad,w1);
		
		hipMemcpy(c+9*n, ad+2, (n-2)*sizeof(float), hipMemcpyDeviceToHost);
		c[10*n-2] = c[10*n-3] + a[10*n-2];
		c[10*n-1] = c[10*n-2] + a[10*n-1];
		a[10*n-1] = c[10*n-1];
				
		hipMemcpy(ad, a+10*n-1, n*sizeof(float), hipMemcpyHostToDevice);
		
		kernelFunc1<<< n>>10 , 1024 >>>(ad,w1);
		kernelFunc1<<< n>>20, 1024 >>> (w1,w2);
		kernelFunc3<<< 1 , n>>20 >>> (w2, n>>20);
		
		kernelFunc2<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc2<<< n>>10 , 1024 >>> (ad,w1);
		
		hipMemcpy(c+10*n, ad+2, (n-2)*sizeof(float), hipMemcpyDeviceToHost);
		c[11*n-2] = c[11*n-3] + a[11*n-2];
		c[11*n-1] = c[11*n-2] + a[11*n-1];
		a[11*n-1] = c[11*n-1];
				
		hipMemcpy(ad, a+11*n-1, n*sizeof(float), hipMemcpyHostToDevice);
		
		kernelFunc1<<< n>>10 , 1024 >>>(ad,w1);
		kernelFunc1<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc3<<< 1 , n>>20 >>> (w2, n>>20);
		
		kernelFunc2<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc2<<< n>>10 , 1024 >>> (ad,w1);
		
		hipMemcpy(c+11*n, ad+2, (n-2)*sizeof(float), hipMemcpyDeviceToHost);
		c[12*n-2] = c[12*n-3] + a[12*n-2];
		c[12*n-1] = c[12*n-2] + a[12*n-1];
		a[12*n-1] = c[12*n-1];
				
		hipMemcpy(ad, a+12*n-1, n*sizeof(float), hipMemcpyHostToDevice);
		
		kernelFunc1<<< n>>10 , 1024 >>>(ad,w1);
		kernelFunc1<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc3<<< 1 , n>>20 >>> (w2, n>>20);
		
		kernelFunc2<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc2<<< n>>10 , 1024 >>> (ad,w1);
		
		hipMemcpy(c+12*n, ad+2, (n-2)*sizeof(float), hipMemcpyDeviceToHost);
		c[13*n-2] = c[13*n-3] + a[13*n-2];
		c[13*n-1] = c[13*n-2] + a[13*n-1];
		a[13*n-1] = c[13*n-1];
				
		hipMemcpy(ad, a+13*n-1, n*sizeof(float), hipMemcpyHostToDevice);
		
		kernelFunc1<<< n>>10 , 1024 >>>(ad,w1);
		kernelFunc1<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc3<<< 1 , n>>20 >>> (w2, n>>20);
		
		kernelFunc2<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc2<<< n>>10 , 1024 >>> (ad,w1);
		
		hipMemcpy(c+13*n, ad+2, (n-2)*sizeof(float), hipMemcpyDeviceToHost);
		c[14*n-2] = c[14*n-3] + a[14*n-2];
		c[14*n-1] = c[14*n-2] + a[14*n-1];
		a[14*n-1] = c[14*n-1];
				
		hipMemcpy(ad, a+14*n-1, n*sizeof(float), hipMemcpyHostToDevice);
		
		kernelFunc1<<< n>>10 , 1024 >>>(ad,w1);
		kernelFunc1<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc3<<< 1 , n>>20 >>> (w2, n>>20);
		
		kernelFunc2<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc2<<< n>>10 , 1024 >>> (ad,w1);
		
		hipMemcpy(c+14*n, ad+2, (n-2)*sizeof(float), hipMemcpyDeviceToHost);
		c[15*n-2] = c[15*n-3] + a[15*n-2];
		c[15*n-1] = c[15*n-2] + a[15*n-1];
		a[15*n-1] = c[15*n-1];
		
		hipMemcpy(ad, a+15*n-1, n*sizeof(float), hipMemcpyHostToDevice);
		
		kernelFunc1<<< n>>10 , 1024 >>>(ad,w1);
		kernelFunc1<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc3<<< 1 , n>>20 >>> (w2, n>>20);
		
		kernelFunc2<<< n>>20 , 1024 >>> (w1,w2);
		kernelFunc2<<< n>>10 , 1024 >>> (ad,w1);
		
		hipMemcpy(c+15*n, ad+2, (n-2)*sizeof(float), hipMemcpyDeviceToHost);
		c[16*n-2] = c[16*n-3] + a[16*n-2];
		c[16*n-1] = c[16*n-2] + a[16*n-1];
				
		hipFree(ad);
		hipFree(w1);
		hipFree(w2);
	}
}